#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
      printf("hipGetDeviceCount returned %d (%s)\n", error_id, hipGetErrorString(error_id));
      exit(EXIT_FAILURE);
    }
    printf("deviceCount: %d\n", deviceCount);

    int dev, driverVersion = 0, runtimeVersion = 0;
    for (dev= 0; dev < deviceCount; ++dev) {
      hipSetDevice(dev);

      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, dev);

      printf("\n");
      printf("device: %d\n", dev);
      printf("name: %s\n", deviceProp.name);

      error_id = hipRuntimeGetVersion(&runtimeVersion);
      if (error_id != hipSuccess) {
        printf("hipRuntimeGetVersion returned %d (%s)\n", error_id, hipGetErrorString(error_id));
      } else {
        printf("runtime version: %d\n", runtimeVersion);
      }

      error_id = hipDriverGetVersion(&driverVersion);
      if (error_id != hipSuccess) {
        printf("hipDriverGetVersion returned %d (%s)\n", error_id, hipGetErrorString(error_id));
      } else {
        printf("driver version: %d\n", driverVersion);
      }
    }
}
